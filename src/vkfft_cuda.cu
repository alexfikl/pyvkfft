#include "hip/hip_runtime.h"
/* PyVkFFT
   (c) 2021- : ESRF-European Synchrotron Radiation Facility
       authors:
         Vincent Favre-Nicolin, favre@esrf.fr
*/

// We use the CUDA backend
#define VKFFT_BACKEND 1

#include <iostream>
#include <fstream>
#include <memory>
#include "vkFFT.h"
typedef float2 Complex;

using namespace std;

extern "C"{
VkFFTConfiguration* make_config(const int, const int, const int, const int, void*, void*, void*,
                                const int, const int, const int);

VkFFTApplication* init_app(const VkFFTConfiguration*);

void fft(VkFFTApplication* app, void*, void*);

void ifft(VkFFTApplication* app, void*, void*);

void free_app(VkFFTApplication* app);

void free_config(VkFFTConfiguration *config);

int test_vkfft_cuda(int);
}

/** Create the VkFFTConfiguration from the array parameters
*
* \param nx, ny, nz: dimensions of the array. The fast axis is x. In the corresponding numpy array,
* this corresponds to a shape of (nz, ny, nx)
* \param fftdim: the dimension of the transform. If nz>1 and fftdim=2, the transform is only made
* on the x and y axes
* \param buffer, buffer_out: pointer to the GPU data source and destination arrays. These
*  can be fake and the actual buffers supplied in fft() and ifft. However buffer should be non-zero,
*  and buffer_out should be non-zero only for an out-of-place transform.
* \param hstream: the stream handle (hipStream_t)
* \param norm: 0, the L2 norm is multiplied by the size on each transform, 1, the inverse transform
*   divides the L2 norm by the size.
* \param precision: number of bits per float, 16=half, 32=single, 64=double precision
* \return: the pointer to the newly created VkFFTConfiguration, or 0 if an error occured
*/
VkFFTConfiguration* make_config(const int nx, const int ny, const int nz, const int fftdim,
                                void *buffer, void *buffer_out, void* hstream,
                                const int norm, const int precision, const int r2c)
{
  VkFFTConfiguration *config = new VkFFTConfiguration({});
  config->FFTdim = fftdim;
  config->size[0] = nx;
  config->size[1] = ny;
  config->size[2] = nz;
  config->normalize = norm;
  config->performR2C = r2c;
  switch(precision)
  {
      case 2 : config->halfPrecision = 1;
      case 8 : config->doublePrecision = 1;
  };

  hipDevice_t *dev = new hipDevice_t;
  if(hstream != 0)
  {
    // Get context then device from current context
    hipCtx_t ctx = nullptr;
    hipError_t res = cuStreamGetCtx ((hipStream_t)hstream, &ctx);
    if(res != hipSuccess)
    {
      cout << "Could not get the current device from given stream"<<endl;
      return 0;
    }
    res = hipCtxPushCurrent (ctx);
    res = hipCtxGetDevice(dev);
    if(res != hipSuccess)
    {
      cout << "Could not get the current device from supplied stream's context."<<endl;
      return 0;
    }
    res = hipCtxPopCurrent (&ctx);

    config->stream = new hipStream_t((hipStream_t) hstream);
    config->num_streams = 1;
  }
  else
  {
    // Get device from current context
    hipError_t res = hipCtxGetDevice(dev);
    if(res != hipSuccess)
    {
      cout << "Could not get the current device. Was a CUDA context created ?"<<endl;
      return 0;
    }
  }
  config->device = dev;

  void ** pbuf = new void*;
  *pbuf = buffer;

  uint64_t* psize = new uint64_t;
  if(r2c) *psize = (uint64_t)((nx / 2 + 1) * ny * nz * precision * 2);
  else *psize = (uint64_t)(nx * ny * nz * precision * 2);
  config->bufferSize = psize;

  if(buffer_out != NULL)
  {
    // Calculations are made in buffer, so with buffer != inputBuffer we keep the original data
    void ** pbufout = new void*;
    *pbufout = buffer_out;

    config->buffer = pbufout;
    config->inputBuffer = pbuf;

    config->inputBufferSize = psize;

    config->isInputFormatted = 1;
  }
  else
  {
    config->buffer = pbuf;
  }


  /*
  cout << "make_config: "<<config<<" "<<endl<< config->buffer<<", "<< *(config->buffer)<<", "
       << config->size[0] << " " << config->size[1] << " " << config->size[2] << " "<< config->FFTdim
       << " " << *(config->bufferSize) << endl;
  */
  return config;
}

/** Initialise the VkFFTApplication from the given configuration.
*
* \param config: the pointer to the VkFFTConfiguration
* \return: the pointer to the newly created VkFFTApplication
*/
VkFFTApplication* init_app(const VkFFTConfiguration* config)
{
  VkFFTApplication* app = new VkFFTApplication({});
  const int res = initializeVkFFT(app, *config);
  /*
  cout << "init_app: "<<config<<endl<< config->buffer<<", "<< *(config->buffer)<<", "
       << config->size[0] << " " << config->size[1] << " " << config->size[2] << " "<< config->FFTdim
       << " " << *(config->bufferSize) << endl<<endl;
  cout<<res<<endl<<endl;
  */
  if(res!=0)
  {
    cout << "VkFFTApplication initialisation failed: " << res << endl;
    delete app;
    return 0;
  }
  return app;
}

void fft(VkFFTApplication* app, void *in, void *out)
{
  *(app->configuration.buffer) = out;  // unnnecessary ?
  *(app->configuration.inputBuffer) = in;
  *(app->configuration.outputBuffer) = out;
  VkFFTAppend(app, -1, NULL);
}

void ifft(VkFFTApplication* app, void *in, void *out)
{
  *(app->configuration.buffer) = out;  // unnnecessary ?
  *(app->configuration.inputBuffer) = in;
  *(app->configuration.outputBuffer) = out;
  VkFFTAppend(app, 1, NULL);
}

/** Free memory allocated during make_config()
*
*/
void free_app(VkFFTApplication* app)
{
  if(app != NULL)
  {
    deleteVkFFT(app);
    free(app);
  }
}

/** Free memory associated to the vkFFT app
*
*/
void free_config(VkFFTConfiguration *config)
{
  free(config->device);
  // Only frees the pointer to the buffer pointer, not the buffer itself.
  free(config->buffer);
  free(config->bufferSize);
  free(config);
  if(config->isOutputFormatted) free(config->outputBuffer);
  if(config->stream != 0) free(config->stream);
}

/** Basic test function
*
*/
int test_vkfft_cuda(const int size)
{
  printf("vkFFT-cuda is starting...\n");

  // Allocate host arrays
  Complex* arr0 = (Complex*)malloc(sizeof(Complex) * size);
  Complex* arr1 = (Complex*)malloc(sizeof(Complex) * size);
  Complex* arr2 = (Complex*)malloc(sizeof(Complex) * size);

  // Init arrays
  for (unsigned int i = 0; i < size; ++i) {
      arr0[i].x = rand() / (float)RAND_MAX - 0.5f + sin((float)i * 2 * 3.141592653589f / size * 7.7);
      arr0[i].y = 0;
      arr1[i].x = arr0[i].x;
      arr1[i].y = 0;
      arr2[i].x = arr0[i].x;
      arr2[i].y = 0;
  }
  int mem_size = sizeof(Complex) * size;
  // Allocate device memory
  Complex* d1;
  hipMalloc((void**)&d1, mem_size);
  Complex* d2;
  hipMalloc((void**)&d2, mem_size);
  // Copy host memory to device
  hipMemcpy(d1, arr1, mem_size, hipMemcpyHostToDevice);
  hipMemcpy(d2, arr2, mem_size, hipMemcpyHostToDevice);

  //////////////////////////////////////////////// vkFFT ///////////////////////////////////////
  printf("Using vkFFT (CUDA backend)\n");

  // Perform the same FFT using vkFFT
  VkFFTConfiguration configuration = {};

  configuration.FFTdim = 1; //FFT dimension, 1D, 2D or 3D (default 1).
  configuration.size[0] = size; //Multidimensional FFT dimensions sizes (default 1). For best performance (and stability), order dimensions in descendant size order as: x>y>z.
  configuration.size[1] = 1;
  configuration.size[2] = 1;

  // hipInit(0); // Should not be necessary ?
  hipDevice_t* dev = new hipDevice_t;
  hipError_t res = hipCtxGetDevice (dev);
  if(res != hipSuccess)
  {
    cout << "Could not get the current device. Was a CUDA context created ?"<<endl;
    return res;
  }
  configuration.device = dev;

  // hipCtx_t context;
  // hipCtxCreate(&vkGPU->context, 0, vkGPU->device);

  configuration.buffer = (void**)&d2;
  uint64_t bufferSize = mem_size;
  configuration.bufferSize = &bufferSize;

  // Initialize applications. This function loads shaders, creates pipeline and configures FFT based on configuration file. No buffer allocations inside VkFFT library.
  VkFFTApplication app = {};
  uint32_t res1 = initializeVkFFT(&app, configuration);  // TODO
  if(res1!=0)
  {
    free(arr0);
    free(arr1);
    free(arr2);
    hipFree(d1);
    hipFree(d2);
    delete dev;
    cout << "Something went wrong intialising the VkFFTApplication !" << endl;
    return res1;
  }

  VkFFTAppend(&app, -1, NULL);

  hipDeviceSynchronize(); // necessary ?
  deleteVkFFT(&app);

  //////////////////////////////////////////////// vkFFT End ///////////////////////////////////

  // Copy device memory to host
  hipMemcpy(arr1, d1, mem_size, hipMemcpyDeviceToHost);
  hipMemcpy(arr2, d2, mem_size, hipMemcpyDeviceToHost);

  /*
  ofstream out("results.dat");
  for (unsigned int i = 0; i < size; ++i)
  {
      out << arr0[i].x <<"+"<<arr0[i].x<<"j "
          << arr1[i].x <<"+"<<arr1[i].x<<"j "
          << arr2[i].x <<"+"<<arr2[i].x<<"j "<<endl;
  }
  */

  // cleanup memory
  free(arr0);
  free(arr1);
  free(arr2);
  hipFree(d1);
  hipFree(d2);
  delete dev;
  cout << "Finished VkFFT basic test"<<endl;
  return 0;
}
